#include "hip/hip_runtime.h"
// BFSTEST : Test breadth-first search in a graph.
// 
// example: cat sample.txt | ./bfstest 1
//
// John R. Gilbert, 17 Feb 20ll

#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <math.h>
#include <time.h>
#include <unistd.h>
#include <cutil_inline.h>
#define VISITED 1
#define UNVISITED 0
#define VTXNUM 10
#define EDGENUM 17 
#define conti 0
#define stop 1

int* push_back(int*, int*, int*);

/* global state */
struct timespec  start_time;                                 
struct timespec  end_time;  

int* vtx[VTXNUM + 1];
int vector_pos[VTXNUM + 1];
int level[VTXNUM + 1];
int VISITED_CHECK[VTXNUM + 1];
int nbr_list[EDGENUM];
int nbr_offset[VTXNUM + 1];


int nv, ne = 0;


int* d_nbr_list;
int* d_nbr_offset;
int* d_level;
int* d_VISITED_CHECK;
int* d_lvl;
int* d_q2;
int* d_vtx_offset;
int* d_vtx_size;
int* d_num_block;
int* d_q2_size;
int* d_count;

int threadsPerBlock = 256;
int blocksPerGrid;


unsigned int seed = 0x12345678;
unsigned int myrand(unsigned int *seed, unsigned int input) {  
	*seed = (*seed << 13) ^ (*seed >> 15) + input + 0xa174de3;
	return *seed;
};

void sig_check(int nv, int* level) {    
	int i;
	unsigned int sig = 0x123456;

	for(i = 0; i < nv; i++)
	{    
		myrand(&sig, level[i]);    
	}           

	printf("Computed check sum signature:0x%08x\n", sig);
	if(sig == 0x18169857)
		printf("Result check of sample.txt by signature successful!!\n");
	else if(sig == 0xef872cf0)
		printf("Result check of TEST1 by signature successful!!\n");
	else if(sig == 0xe61d1d00) 
		printf("Result check of TEST2 by signature successful!!\n");
	else if(sig == 0x29c12a44)
		printf("Result check of TEST3 by signature successful!!\n");
	else
		printf("Result check by signature failed!!\n");
}

__device__ void enqueue_local(int* index, int nbr, int* q_local)
{
	int old_index = atomicAdd(index, 1);
	q_local[old_index] = nbr;
}

__device__ void global_barrier(int private_num_block, int* count)
{
	__syncthreads();
	if(threadIdx.x == 0){
		atomicAdd(count, 1);
		while(*count < private_num_block){
			;
		}
	}
	__syncthreads();
}


///////////test kernel function//////////////
__global__ void test(int* global_mem_size)
{
	*global_mem_size = 200;
}




__global__ void bfs_kernel(int* q2, int* q2_size, int* vtx_offset, int* vtx_size, int* nbr_list,int* level, int* VISITED_CHECK, int* num_block, int* count)
{
	__shared__ int q_local[1024];
	__shared__ int index;
	int private_num_block = (*num_block);
	if(threadIdx.x == 0)index = 0;
	int i, j= 0;
	int idx = blockDim.x*blockIdx.x + threadIdx.x;
	int offset = vtx_offset[idx];
	for(i = 0; i < vtx_size[idx]; i++){
		int nbr = nbr_list[offset + i];
		int old_status = atomicExch(&VISITED_CHECK[nbr], VISITED);
		if(old_status == UNVISITED){
			//enqueue_local(&index, nbr, q_local);

			//////////TEST///////////////
			q_local[j] = nbr;
			j++;
			//////////////////////////
		}
	}
//	global_barrier(private_num_block, count);
	
	if(threadIdx.x == 0)
		atomicAdd(q2_size, j);
	
//	global_barrier(private_num_block, count);
	
	if(threadIdx.x == 0){
		for(i = 0; i < *q2_size; i++)
			q2[i] = q_local[i];
	}
	
}


void read_edge_list (int** vtx, int* vector_pos, int* level) {
	int max_edges = 100000000;
	int nedges, nr, t, h, max;
	nedges = 0;
	nr = scanf("%i %i",&h,&t);
	if(t > h)	nv = t;
	else	nv = h;
	while (nr == 2) {
		if (nedges >= max_edges) {
			printf("Limit of %d edges exceeded.\n",max_edges);
			exit(1);
		}
		vtx[h] = push_back(vtx[h], &t, &vector_pos[h]);
		level[h] = -1;
		level[t] = -1;
		ne++;
		if(t > h)	max = t;
		else	max = h;
		if(max > nv)	nv = max;
		nr = scanf("%i %i",&h,&t);
	}
}

void init_nbr_list(int** vtx, int* vector_pos, int* nbr_list, int* nbr_offset)
{
	int i, j, e;
	e = 0;
	nbr_offset[0] = 0;
	for(j = 0; j < vector_pos[0]; j++){
			nbr_list[e] = vtx[i][j];
			e++;
	}
	for(i = 1; i < VTXNUM; i++){
		nbr_offset[i] = vector_pos[i - 1] + nbr_offset[i - 1];
		for(j = 0; j < vector_pos[i]; j++){
			nbr_list[e] = vtx[i][j];
			e++;
		}
		//free(vector[i]);
		//free(vtx[i]);
	}
	//free(vtx);
}

void bfs()
{
	int i, q1_size, q2_size;
	int init_value = 0;
	int* q_tmp;
	int* vtx_offset;
	int* vtx_size;
	cutilSafeCall(hipMalloc((void**)&d_q2, EDGENUM*sizeof(int)));
	cutilSafeCall(hipMalloc((void**)&d_nbr_list, EDGENUM*sizeof(int)));
	cutilSafeCall(hipMalloc((void**)&d_level, (VTXNUM + 1)*sizeof(int)));
	cutilSafeCall(hipMalloc((void**)&d_VISITED_CHECK, (VTXNUM + 1)*sizeof(int)));
	cutilSafeCall(hipMalloc((void**)&d_q2_size, sizeof(int)));
	cutilSafeCall(hipMalloc((void**)&d_count, sizeof(int)));
	cutilSafeCall(hipMemcpy(d_nbr_list, nbr_list, (nv + 1)*sizeof(int), hipMemcpyHostToDevice));
	cutilSafeCall(hipMemcpy(d_level, level, (nv + 1)*sizeof(int), hipMemcpyHostToDevice));
	cutilSafeCall(hipMemcpy(d_VISITED_CHECK, VISITED_CHECK, (nv + 1)*sizeof(int), hipMemcpyHostToDevice));


//////////////////////////FOR LOOP////////////////////////////

	cutilSafeCall(hipMemcpy(d_count, &init_value, sizeof(int), hipMemcpyHostToDevice));
	cutilSafeCall(hipMemcpy(d_q2_size, &init_value, sizeof(int), hipMemcpyHostToDevice));
	q1_size = 1;
	q_tmp = (int*)realloc(NULL, q1_size*sizeof(int));
	vtx_offset = (int*)realloc(NULL, q1_size*sizeof(int));
	vtx_size = (int*)realloc(NULL, q1_size*sizeof(int));
	for(i = 0; i < q1_size; i++){
		q_tmp[i] = 1;
		vtx_offset[i] = nbr_offset[q_tmp[i]];
		vtx_size[i] = vector_pos[q_tmp[i]];
	}
	cutilSafeCall(hipMalloc((void**)&d_vtx_offset, q1_size*sizeof(int)));
	cutilSafeCall(hipMalloc((void**)&d_vtx_size, q1_size*sizeof(int)));
	cutilSafeCall(hipMemcpy(d_vtx_offset, vtx_offset, q1_size*sizeof(int), hipMemcpyHostToDevice));
	cutilSafeCall(hipMemcpy(d_vtx_size, vtx_size, q1_size*sizeof(int), hipMemcpyHostToDevice));

	if(q1_size < 256){
		threadsPerBlock = q1_size;
		blocksPerGrid = 1;
	}
	else{
		threadsPerBlock = 256;
		blocksPerGrid = q1_size/threadsPerBlock;
	}
	cutilSafeCall(hipMalloc((void**)&d_num_block, sizeof(int)));
	cutilSafeCall(hipMemcpy(d_num_block, &blocksPerGrid, sizeof(int), hipMemcpyHostToDevice));
	bfs_kernel<<<threadsPerBlock, blocksPerGrid>>>(d_q2, d_q2_size, d_vtx_offset, d_vtx_size, d_nbr_list, d_level, d_VISITED_CHECK, d_num_block, d_count);
	cutilCheckMsg("kernel launch failure");




///////test/////////////////////////////////////
/*	int* global_mem_size;
	cutilSafeCall(hipMalloc((void**)&global_mem_size, sizeof(int)));
	cutilSafeCall(hipMemcpy(global_mem_size, &q1_size, sizeof(int), hipMemcpyHostToDevice));

	test<<<1, 1>>>(global_mem_size);
	cutilCheckMsg("kernel launch failure");

	cutilSafeCall(hipMemcpy(&q2_size, global_mem_size, sizeof(int), hipMemcpyDeviceToHost));*/
//////////////////////////////////////



	cutilSafeCall(hipMemcpy(&q2_size, d_q2_size, sizeof(int), hipMemcpyDeviceToHost));
	printf("q2_size: %d\n", q2_size);
	
	q_tmp = (int*)realloc(NULL, q2_size*sizeof(int));

	cutilSafeCall(hipMemcpy(q_tmp, d_q2, q2_size*sizeof(int), hipMemcpyDeviceToHost));


	////////////////////////TEST RESULT///////////////////////
	printf("The vtx of q2: ");
	for(i = 0; i < q2_size ; i++)
		printf("%d ", q_tmp[i]);
	printf("\n");

}


int main (int argc, char* argv[]) {
	int startvtx;
	int i /*j*/;
	/*if (argc == 2) {
		startvtx = atoi (argv[1]);
	} else {
		printf("usage:   bfstest <startvtx> < <edgelistfile>\n");
		printf("example: cat sample.txt | ./bfstest 1\n");
		exit(1);
	}*/
	startvtx = 1;

	//int** vtx = (int**)malloc((VTXNUM + 1)*sizeof(int*));
	for(i = 0; i <= VTXNUM; i++){
		if(vtx[i] == NULL)
			vtx[i] = (int*)malloc(sizeof(int));
	}
	//int* nbr_list = (int*)malloc(EDGENUM*sizeof(int));
	//int* vector_pos = (int*)malloc((VTXNUM + 1)*sizeof(int));
	//int* level = (int*)malloc((VTXNUM + 1)*sizeof(int));
	read_edge_list(vtx, vector_pos, level);
	nv++;
	printf("Num of Edges: %d\n", ne);
	printf("Num of Vertex: %d\n", nv);
	printf("Num of Vertex[1]'s link: %d\n", vector_pos[1]);


	//Print the Info of eacg vertex//
	/*for(i = 0; i < nv; i++){
		printf("Vertex[%d]: ", i);
		j = 1;
		while(j<=vector_pos[i]){
			printf("%d ", vtx[i][j]);
			j++;
		}
		printf("\nNum of link: %d\n", vector_pos[i]);
	}*/

	//int* nbr_offset = (int*)malloc((VTXNUM + 1)*sizeof(int));
	//int* VISITED_CHECK = (int*)malloc((VTXNUM + 1)*sizeof(int));

	clock_gettime(CLOCK_REALTIME, &start_time); //stdio scanf ended, timer starts  //Don't remove it

	init_nbr_list(vtx, vector_pos, nbr_list, nbr_offset);

	// Print all the neighbors
	for(i = 0; i < 17; i++)
		printf("%d ", nbr_list[i]);
	printf("\n");
	for(i = 0; i < 10; i++)
		printf("%d ", nbr_offset[i]);
	printf("\n");

	bfs();

		//Print the level of each vertex//
	//for(i = 0; i < 10; i++)
	//	printf("The level of Vertex[%d]: %d\n", i, level[i]);
	//for(i = 0; i < 10; i++)
	//	printf("The VISITED of Vertex[%d]: %d\n", i, VISITED_CHECK[i]);


	clock_gettime(CLOCK_REALTIME, &end_time);  //graph construction and bfs completed timer ends  //Don't remove it


	printf("Starting vertex for BFS is %d.\n\n",startvtx);

	//Don't remove it
	printf("s_time.tv_sec:%ld, s_time.tv_nsec:%09ld\n", start_time.tv_sec, start_time.tv_nsec);
	printf("e_time.tv_sec:%ld, e_time.tv_nsec:%09ld\n", end_time.tv_sec, end_time.tv_nsec);
	if(end_time.tv_nsec > start_time.tv_nsec)
	{
		printf("[diff_time:%ld.%09ld sec]\n",
				end_time.tv_sec - start_time.tv_sec,
				end_time.tv_nsec - start_time.tv_nsec);
	}
	else
	{
		printf("[diff_time:%ld.%09ld sec]\n",
				end_time.tv_sec - start_time.tv_sec - 1,
				end_time.tv_nsec - start_time.tv_nsec + 1000*1000*1000);
	}
	sig_check(nv, level);

	return 0;
}


int* push_back(int *array, int* data, int* pos)
{
	if((*pos) == 0){
//		array = (int*)malloc(sizeof(int));
		array[0] = (*data);
		(*pos) = 1;
	}
	else{
		array = (int*)realloc(array, (*pos+1)*sizeof(int));
		array[(*pos)] = (*data);
		(*pos)++;
	}
	return array;
}


